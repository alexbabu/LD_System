#include "hip/hip_runtime.h"
/*
 * CPreprocessing.cpp
 *
 *  Created on: Apr 16, 2016
 *      Author: karathra
 */

#include "CPreprocessing.h"
//#incldue "Parallel.cu"

CPreprocessing::CPreprocessing(char* pcBuf)
{
	// TODO Auto-generated constructor stub
	m_matImage = imread(pcBuf, 1);

	m_matLeftROIImage = m_matImage(Rect(LEFT_ROI_STARTX, LEFT_ROI_STARTY,
			LEFT_ROI_WIDTH, LEFT_ROI_HIEGHT));
	m_matRightROIImage = m_matImage(Rect(RIGHT_ROI_STARTX, RIGHT_ROI_STARTY,
			RIGHT_ROI_WIDTH, RIGHT_ROI_HIEGHT));
	RGBtoGray(LEFT);
}

__global__ void cudaPrepareHough(unsigned char* d_Hough,
		unsigned char* d_GrayROIImage,
		int nRows,
		int nCols,
		int nMaxRho)
{
	__shared__ int
	matTileNormalisation[NOOFELEMENTSBEFORESOBEL][NOOFELEMENTSBEFORESOBEL];
	__shared__ int
	matTileResultAverage[NOOFELEMENTSBEFORESOBEL][NOOFELEMENTSBEFORESOBEL];

	int nThreadRow;
	int nThreadCol;
	int nIndex;

	int nTileRow;
	int nTileCol;

	nThreadRow = blockIdx.x * blockDim.x + threadIdx.x;
	nThreadCol = blockIdx.y * blockDim.y + threadIdx.y;
	nIndex = nThreadRow * nCols + nThreadCol;

	nTileRow = threadIdx.x;
	nTileCol = threadIdx.y;

	if(nTileRow == 0)
	{
		//checking for top left corner
	    if(nTileCol == 0)
	    {
	    	//if top left corner, thread should copy a[i - 1][j - 1],
	    	//a[i - 1][j],a[i][j] and a[i][j - 1]
	    	matTileNormalisation[0][0] =
	    			d_GrayROIImage[nIndex - nCols - 1];
	    	matTileNormalisation[0][1] =
	    			d_GrayROIImage[nIndex - nCols];
	    	matTileNormalisation[1][1] =
	    			d_GrayROIImage[nIndex];
	    	matTileNormalisation[1][0] =
	    			d_GrayROIImage[nIndex - 1];
	    }
	    else if(nTileCol == NO_OF_THREADS - 1)
	    {
	    	//if top right corner, thread should copy a[i - 1][j + 1],
	    	//a[i - 1][j],a[i][j] and a[i][j + 1]
	    	matTileNormalisation[0][nTileCol + 2] =
	    			d_GrayROIImage[nIndex - nCols  + 1];
	    	matTileNormalisation[0][nTileCol + 1] =
	    			d_GrayROIImage[nIndex - nCols];
	    	matTileNormalisation[1][nTileCol + 1] =
	    			d_GrayROIImage[nIndex];
	    	matTileNormalisation[1][nTileCol + 2] =
	    			d_GrayROIImage[nIndex + 1];

	    }
	    else
	    {
	    	//other threads of first row should copy a[i - 1][j] and a[i][j]
	        matTileNormalisation[0][nTileCol + 1] =
	        		d_GrayROIImage[nIndex - nCols];
	        matTileNormalisation[1][nTileCol + 1] =
	        		d_GrayROIImage[nIndex];
	    }
	}
	else if(nTileRow == NO_OF_THREADS - 1)
	{
		if(nTileCol == 0)
	    {
			//if bottom left corner, thread should copy a[i + 1][j - 1], a[i + 1][j],
			//a[i][j] and a[i][j - 1]
	        matTileNormalisation[nTileRow + 2][0] =
	        		d_GrayROIImage[nIndex + nCols - 1];
	        matTileNormalisation[nTileRow + 2][1] =
	        		d_GrayROIImage[nIndex + nCols];
	        matTileNormalisation[nTileRow + 1][1] =
	        		d_GrayROIImage[nIndex];
	        matTileNormalisation[nTileRow + 1][0] =
	        		d_GrayROIImage[nIndex - 1];
	     }
	     else if(nTileCol == NO_OF_THREADS - 1)
	     {
	    	 //if bottom right corner, thread should copy a[i + 1][j + 1],
	    	 //a[i + 1][j],a[i][j] and a[i][j + 1]
	         matTileNormalisation[nTileRow + 2][nTileCol + 2] =
	        		 d_GrayROIImage[nIndex + nCols  + 1];
	         matTileNormalisation[nTileRow + 2][nTileCol + 1] =
	        		 d_GrayROIImage[nIndex + nCols];
	         matTileNormalisation[nTileRow + 1][nTileCol + 1] =
	        		 d_GrayROIImage[nIndex];
	         matTileNormalisation[nTileRow + 1][nTileCol + 2] =
	        		 d_GrayROIImage[nIndex + 1];

	     }
	     else
	     {
	    	 //other threads of the bottom row are supposed to copy
	    	 //a[i + 1][j] and a[i][j]
	         matTileNormalisation[nTileRow + 2][nTileCol + 1] =
	        		 d_GrayROIImage[nIndex + nCols];
	         matTileNormalisation[nTileRow + 1][nTileCol + 1] =
	        		 d_GrayROIImage[nIndex];
	     }
	 }
	 else if(nTileCol == 0)
	 {
		 //threads of first column are supposed to copy a[i][j - 1] and a[i][j]
	     matTileNormalisation[nTileRow + 1][0] = d_GrayROIImage[nIndex - 1];
	     matTileNormalisation[nTileRow + 1][1] = d_GrayROIImage[nIndex];
	  }
	  else if(nTileCol == NO_OF_THREADS - 1)
	  {
		  //threads of last column are supposed to copy a[i][j + 1] and a[i][j]
	      matTileNormalisation[nTileRow + 1][nTileCol + 2] =
	    		  d_GrayROIImage[nIndex + 1];
	      matTileNormalisation[nTileRow + 1][nTileCol + 1] =
	    		  d_GrayROIImage[nIndex];
	  }
	  else
	  {
		  //rest of the threads copy a[i][j]
	      matTileNormalisation[nTileRow + 1][nTileCol + 1] = d_GrayROIImage[nIndex];
	  }
	  __syncthreads();

	  //normalisation starts
	  //3 * 3 kernel of top hat
	  float nTempSum;
	  nTempSum = (matTileNormalisation[nTileRow][nTileCol] +
			  matTileNormalisation[nTileRow][nTileCol + 1] +
			  matTileNormalisation[nTileRow][nTileCol + 2] +
			  matTileNormalisation[nTileRow + 1][nTileCol] +
			  matTileNormalisation[nTileRow + 1][nTileCol + 1] +
			  matTileNormalisation[nTileRow + 1][nTileCol + 2] +
			  matTileNormalisation[nTileRow + 2][nTileCol] +
			  matTileNormalisation[nTileRow + 2][nTileCol + 1] +
			  matTileNormalisation[nTileRow + 2][nTileCol + 2]) / 9;

	  //get the Normalisation threshold into registry
	  int nNormalisationThresh = 160;

	  if((int)nTempSum > nNormalisationThresh)
	  {
		  matTileResultAverage[nTileRow + 1][nTileCol + 1] = 255;
	  }
	  else
	  {
		  matTileResultAverage[nTileRow + 1][nTileCol + 1] = 0;
	  }

	  __syncthreads();

	  __shared__ float
	  matTileSobel[NOOFELEMENTSBEFORESOBEL - 2][NOOFELEMENTSBEFORESOBEL - 2];
	  int nTempSobelSum;
	  float nSobelx;
	  float nSobely;
	  float nEdgeMag;
	  //start the Sobel edge detector
	  nTempSobelSum = matTileResultAverage[nTileRow][nTileCol + 2] -
			  matTileResultAverage[nTileRow + 2][nTileCol];
	  nSobelx = nTempSobelSum +
			  matTileResultAverage[nTileRow + 2][nTileCol + 2] -
			  matTileResultAverage[nTileRow][nTileCol] +
              2 * (matTileResultAverage[nTileRow + 1][nTileCol + 2] -
              matTileResultAverage[nTileRow + 1][nTileCol]);
	  nSobely = nTempSobelSum +
			  matTileResultAverage[nTileRow][nTileCol] -
			  matTileResultAverage[nTileRow + 2][nTileCol + 2] +
	          2 * (matTileResultAverage[nTileRow][nTileCol + 1] -
	          matTileResultAverage[nTileRow + 2][nTileCol + 1]);

	  nEdgeMag = sqrt(nSobelx * nSobelx + nSobely * nSobely);

	  int nSobelThresh = 50;

	  if((int)nEdgeMag > nSobelThresh)
	  {
		  matTileSobel[nTileRow][nTileCol] = 255;
	  }
	  else
	  {
		  matTileSobel[nTileRow][nTileCol] = 0;
	  }

	  //d_SobelImage[nThreadRow * (nCols - 2) + nThreadCol] =
			  //matTileSobel[nTileRow][nTileCol];

	  __shared__ int matHough[NO_OF_RHO][NO_OF_THETTA];

	  float nThettaCnt;
	  float nTempAngle = (PI / 180);
	  int nRho;
	  nRows = nRows - 2;
	  nCols = nCols - 2;
	  nIndex = nThreadRow * nCols + nThreadCol;
	  int nXTerm = nThreadCol - nCols;
	  int nYTerm = nThreadRow - nRows;
	  //int nMaxRho = ceil(sqrt(pow(nRows, 2) + pow(nCols, 2)));

	  if(nIndex < 200)
	  {
		  matHough[nThreadRow][nThreadCol] = 0;
	  }
	  if(matTileSobel[nTileRow][nTileCol])
	  {
		  for(nThettaCnt = LEFT_MIN_THETTA; nThettaCnt < LEFT_MAX_THETTA; nThettaCnt++)
		  {
			  nRho = ceil((float)nXTerm * cos(nTempAngle * nThettaCnt) +
					  (float)nYTerm * sin(nTempAngle * nThettaCnt));

			  nRho = nRho + nMaxRho;
			  //printf("%d")
			  if(nRho < LEFT_MAX_RHO && nRho > LEFT_MIN_RHO)
			  {
				  nRho = nRho - LEFT_MIN_RHO;
				  int nThetta = nThettaCnt - LEFT_MIN_THETTA;
				  atomicAdd((*(matHough + nRho) + nThetta), 1);
			  }
		  }
	  }

	  __syncthreads();

	  if(nIndex < 200)
	  {
		  atomicAdd((int*)(d_Hough + nIndex), *((int*)matHough + nIndex));
	  }
}

void CPreprocessing::RGBtoGray(int nSide)
{
	int nRows = m_matLeftROIImage.rows;
	int nCols = m_matLeftROIImage.cols;
	//int nColorSize = nRows * nCols * 3;
	int nGraySize = nRows * nCols;
	int nSobelSize = (nRows -2) * (nCols - 2);
	int nMaxRho = ceil(sqrt(pow(nRows -2, 2) + pow(nCols - 2, 2)));

	//unsigned char* d_ROIImageB;
	//unsigned char* d_ROIImageG;
	//unsigned char* d_ROIImageR;
	unsigned char* d_GrayROIImage;
	unsigned char* d_Hough;
	//unsigned char* d_SobelImage;

	//vector<Mat> vecmatChannels;

	//split(m_matLeftROIImage, vecmatChannels);
	//m_matLeftROIGray = Mat(nRows, nCols, CV_8UC1);
	//m_matLeftSobel = Mat(nRows - 2, nCols - 2, CV_8UC1);
	m_matLeftHough = Mat(100, 25, CV_8UC1);
	cvtColor(m_matLeftROIImage, m_matLeftGrayImage, CV_BGR2GRAY);

	//hipMalloc((void**)&d_ROIImage, nColorSize);
	//hipMalloc((void**)&d_ROIImageB, nGraySize);
	//hipMalloc((void**)&d_ROIImageG, nGraySize);
	//hipMalloc((void**)&d_ROIImageR, nGraySize);
	hipMalloc((void**)&d_GrayROIImage, nGraySize);
	hipMalloc((void**)&d_Hough, 100 * 20);
	//hipMalloc((void**)&d_SobelImage, nSobelSize);

	//imwrite("ROIImage.jpg", m_matLeftROIImage);

	//hipMemcpy(d_ROIImageB, vecmatChannels[0].data, nGraySize,
	//hipMemcpyHostToDevice);
	//hipMemcpy(d_ROIImageG, vecmatChannels[1].data, nGraySize,
	//hipMemcpyHostToDevice);
	//hipMemcpy(d_ROIImageR, vecmatChannels[2].data, nGraySize,
	//hipMemcpyHostToDevice);
	hipMemcpy(d_GrayROIImage, m_matLeftGrayImage.data,
			nGraySize, hipMemcpyHostToDevice);
	//hipEvent_t start,stop;
	//float elapsed_time;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//hipEventRecord(start,0);

	//for(int nCnt = 0; nCnt <10; nCnt++)
	//{
	//cudaRGBtoGray<<<RGBGRAYGRIDSIZE, RGBGRAYBLOCKSIZE>>>
			//(d_ROIImageB, d_ROIImageG, d_ROIImageR, d_GrayROIImage);

	//hipDeviceSynchronize();

	dim3 dimGrid(HOUGH_X_BLOCK, HOUGH_Y_BLOCK);
	dim3 dimBlock(HOUGH_Y_THREADS, HOUGH_Y_THREADS);

	cudaPrepareHough<<<dimGrid, dimBlock>>>
			(d_Hough, d_GrayROIImage, nRows, nCols, nMaxRho);
	//}

	hipDeviceSynchronize();
	//hipEventRecord(stop);
	//hipEventSynchronize(stop);
	//hipEventElapsedTime(&elapsed_time,start, stop);

	//printf("The operation was successful, time = %2.6f\n",elapsed_time/10);

	hipMemcpy(m_matLeftHough.data, d_Hough, nSobelSize, hipMemcpyDeviceToHost);

	//hipFree(d_ROIImageB);
	//hipFree(d_ROIImageG);
	//hipFree(d_ROIImageR);
	hipFree(d_GrayROIImage);
	hipFree(d_Hough);
	//hipFree(d_SobelImage);

	//vecmatChannels[0].release();
	//vecmatChannels[1].release();
	//vecmatChannels[2].release();
	//printf("buhahha");
	//imwrite("Hough.jpg", m_matLeftHough);
}

CPreprocessing::~CPreprocessing() {
	// TODO Auto-generated destructor stub
	m_matImage.release();
	m_matLeftROIImage.release();
	//m_matRightROIImage.release();
	m_matLeftHough.release();
	//m_matLeftROIGray.release();
	//m_matRightROIGray.release();
}
